
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <stdbool.h>

typedef struct
{
    double * m;
    unsigned columns;
    unsigned rows;
}  matrix_t;


matrix_t * alloc_matrix(unsigned rows, unsigned columns)
{
    matrix_t * res = (matrix_t*) malloc( sizeof(matrix_t) );
    res->m = (double *) calloc(columns * rows, sizeof(double));
    res->columns = columns;
    res->rows = rows;
    return res;
}

void initMatrix(matrix_t *m){
    for (int i=0; i < m->rows; i++){
        for (int j=0; j < m->columns; j++){
            m->m[i * m->columns + j] = 1;
        }
    }
}

__global__
void matrix_dot_kernel(double *m1, double *m2, double *res, int m1rows, int m1columns, int m2columns){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < m1rows && col < m2columns) {
        int idx = col + row * m2columns;
        double var = 0.0;
        for (int ii = 0; ii < m1columns; ii++){
            var += m1[ii + row * m1columns] * m2[col + ii * m2columns];
            }
        res[idx] = var;
    }
}

const int TILE_WIDTH = 16;

__global__
void matrix_mul_kernel2 ( double * M, double * N, double * P, int m1rows, int m1columns, int m2columns )
{
    __shared__ double ds_M[ TILE_WIDTH ][ TILE_WIDTH ];
    __shared__ double ds_N[ TILE_WIDTH ][ TILE_WIDTH ];

    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    int row = blockIdx.y * blockDim.y+ ty;
    int col = blockIdx.x * blockDim.x+ tx;

    double pvalue = 0;

    if (row < m1rows && col < m2columns) {
        for (int p = 0; p < ceil((double)m1columns / TILE_WIDTH) ; p ++) {
            // collaborative loading of M and N tiles into shared memory
            ds_M[ty][tx] = M[ row * m1columns + p * TILE_WIDTH + tx ];
            ds_N[ty][tx] = N[(p * TILE_WIDTH + ty) * m2columns + col ];
            __syncthreads () ;

            for (int i = 0; i < TILE_WIDTH ; ++i) {
                pvalue += ds_M[ty][i] * ds_N[i][tx];
            }
            __syncthreads () ;
        }
    }
    P[ row * m2columns + col ] = pvalue ;
}

__global__
void matrix_mul_kernel (double * A, double * B, double * C, int numARows, int numAColumns, int numBColumns )
{
    __shared__ double ds_M[ TILE_WIDTH ][ TILE_WIDTH ];
    __shared__ double ds_N[ TILE_WIDTH ][ TILE_WIDTH ];

    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    int Row = blockIdx.y * blockDim.y+ ty;
    int Col = blockIdx.x * blockDim.x+ tx;

    double Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
            ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
        else
            ds_M[ty][tx] = 0;
        if (Col < numBColumns && m*TILE_WIDTH+ty < numAColumns)
           ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
        else
           ds_N[ty][tx] = 0;

        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; ++k)
           Pvalue += ds_M[ty][k] * ds_N[k][tx];
        __syncthreads();
    }
    if (Row < numARows && Col < numBColumns)
        C[Row*numBColumns+Col] = Pvalue;
}


void matrix_dot(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->rows)  &&
             (m1->rows == res->rows)    &&
             (m2->columns == res->columns));

    double *d1;
    double *d2;
    double *dres;

    hipMalloc((void **)&d1,  m1->rows * m1->columns * sizeof(double));
    hipMalloc((void **)&d2, m2->rows * m2->columns * sizeof(double));
    hipMalloc((void **)&dres, res->rows * res->columns * sizeof(double));

    hipMemcpy(d1, m1->m, m1->rows * m1->columns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d2, m2->m, m2->rows * m2->columns * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((double)m2->columns) / blockDim.x), ceil(((double)m1->rows) / blockDim.y));

    matrix_mul_kernel <<< gridDim, blockDim >>> (d1, d2, dres, m1->rows, m1->columns, m2->columns);

    hipMemcpy(res->m, dres, res->rows * res->columns * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d1);
    hipFree(d2);
    hipFree(dres);
}

int main(){
    matrix_t *m1 = alloc_matrix(50,100);
    matrix_t *m2 = alloc_matrix(100,50);
    matrix_t *m3 = alloc_matrix(50,50);

    initMatrix(m1);
    initMatrix(m2);

    matrix_dot(m1,m2,m3);

    for (int i =0; i<201; i=i+1){
        printf("m[%d] = %d\n", i, (int)m3->m[i]);
    }


    /*printf("m[0] = %d\n", (int)m3->m[0]);
    printf("m[99] = %d\n", (int)m3->m[99]);
    printf("m[9600] = %d\n", (int)m3->m[9600]);*/
    return 0;
}